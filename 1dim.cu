#include <stdio.h>
#include <hip/hip_runtime.h>

#define INIT 1100
void random(int* x){
	for(int i=0;i<INIT;i++){
		x[i] = rand() % 10;
	}
}

__global__ void kernel(int *a, int *b, int *c){
	int x = blockIdx.x * threadIdx.x + threadIdx.x;
	c[x] = a[x] * b[x];
}

int main(void){
	int a[INIT] = {0};
	int b[INIT] = {0};
	int c[INIT] = {0};
	
	for(int i=0;i<INIT;i++){
		printf("%d * %d = %d\n", a[i], b[i], c[i]);
	}
	printf("\n");
	
	int *GA, *GB, *GC;
	random(a);
	hipMalloc((void**)&GA, INIT*sizeof(int));
	hipMemcpy(GA, a, sizeof(int)*INIT, hipMemcpyHostToDevice);

	random(b);
	hipMalloc((void**)&GB, INIT*sizeof(int));
	hipMemcpy(GB, b, sizeof(int)*INIT, hipMemcpyHostToDevice);
	
	for(int i=0;i<INIT;i++){
		printf("%d * %d = %d\n", a[i], b[i], c[i]);
	}
	printf("\n");
	
	hipMalloc((void**)&GC, INIT*sizeof(int));
	
	kernel<<<1, INIT>>>(GA,GB,GC);
	hipMemcpy(c, GC, sizeof(int)*INIT, hipMemcpyDeviceToHost);
	
	for(int i=0;i<INIT;i++){
		printf("%d:\t %d * %d = %d\n", i, a[i], b[i], c[i]);
	}
	
	hipFree(GA);
	hipFree(GB);
	hipFree(GC);
	
	return 0;
}
