#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>


#define ISLAND 1
#define POPULATION 10
#define FACILITY 6

#define H 3 // BAY height
#define W 2 // BAY width

void shuffle(int* facility);

__global__ void calPosition(int *data, short int *bay, float *position){

  int b=blockIdx.x;       //區塊索引 == ISLAND
  int t=threadIdx.x;      //執行緒索引 == POPULATION
  int n=blockDim.x;       //區塊中包含的執行緒數目 == num of ISLAND
  int x=b*n+t;

  int posit = x * FACILITY;
  int bayposit = x * (FACILITY-1);
  // int posit=b*POPULATION*FACILITY+t*FACILITY;            //執行緒在陣列中對應的位置
  // int posofposit = b*POPULATION*FACILITY*2+t*FACILITY*2;

  for(int i=0;i<ISLAND*POPULATION*FACILITY*2;i++){
    position[i] = 0;
  }


			int len = 1;
			int next = 0;
			for(int f=0;f<FACILITY;f++){
				if(bay[bayposit+f] == 0){
					len = len + 1;
				}
				if(bay[bayposit+f] == 1 || f == FACILITY - 1 ){
					if(f == FACILITY - 1 && bay[bayposit+f] == 0){
						len = len - 1;
					}
					float x = W / 2.0 + next;

					for(int j=0;j<len;j++){

						position[posit*2+(f+j-len+1)*2] = x;

						float y = H / (len * 2.0) * ( (j * 2) + 1) ;

						position[posit*2+(f+j-len+1)*2+1] = y;
					}
					len = 1;

					next = next + W;
				}
			}
}

__global__ void calDistance(int *data, float *position, float *distance){

  int b=blockIdx.x;       //區塊索引 == ISLAND
  int t=threadIdx.x;      //執行緒索引 == POPULATION
  int n=blockDim.x;       //區塊中包含的執行緒數目 == num of ISLAND
  int x=b*n+t;

  int posit = x * FACILITY;

  // int posit=b*POPULATION*FACILITY+t*FACILITY;            //執行緒在陣列中對應的位置
  // int posofposit = b*POPULATION*FACILITY*2+t*FACILITY*2;

  for(int i=0;i<ISLAND*POPULATION*FACILITY*FACILITY;i++){
    distance[i] = 0;
  }


  for(int f=0;f<FACILITY;f++){
    // printf("\ndistance calculate facility%d\n", f);
    for(int j=f+1;j<FACILITY;j++){

      float x1 = position[ (posit + f)*2 ];
      float y1 = position[ (posit + f)*2 + 1];

      int x = data[ posit + f ];
      // printf("x = %d\n", x);
      float x2 = position[ (posit + j)*2 ];
      float y2 = position[ (posit + j)*2 + 1];
      int y = data[ posit + j ];
      // printf("y= %d\n", y);
      if(y2 > y1){
        distance[ (posit + x)*FACILITY + y] = sqrt( (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) ) ;
        distance[ (posit + y)*FACILITY + x] = sqrt( (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) ) ;
      }
      else{
        distance[ (posit + x)*FACILITY + y] = sqrt( (x2 - x1) * (x2 - x1) + (y1 - y2) * (y1 - y2) ) ;
        distance[ (posit + y)*FACILITY + x] = sqrt( (x2 - x1) * (x2 - x1) + (y1 - y2) * (y1 - y2) ) ;
      }
    }
  }

}

__global__ void calTotalcost(float *distance, int *cost, float *totalCost){

  int b=blockIdx.x;       //區塊索引 == ISLAND
  int t=threadIdx.x;      //執行緒索引 == POPULATION
  int n=blockDim.x;       //區塊中包含的執行緒數目 == num of ISLAND
  int x=b*n+t;

  int posit = x * FACILITY;

  // int posit=b*POPULATION*FACILITY+t*FACILITY;            //執行緒在陣列中對應的位置
  // int posofposit = b*POPULATION*FACILITY*2+t*FACILITY*2;

  // for(int i=0;i<ISLAND*POPULATION*FACILITY*FACILITY;i++){
  //   totalCost[i] = 0;
  // }


	for(int f=0;f<FACILITY;f++){
		for(int j=0;j<FACILITY;j++){
			totalCost[ (posit + f)*FACILITY + j] = cost[f*FACILITY + j] * distance[ (posit + f)*FACILITY + j];
		}
	}

}


int main(){
  srand(time(NULL));

  int data[ISLAND*POPULATION*FACILITY];
  short int bay[ISLAND*POPULATION*(FACILITY-1)]; //bay

  int facility[FACILITY];

  for(int i=0;i<ISLAND;i++){ // shuffle the sorted facility
		printf("new island%d\n", i);
		for(int p=0;p<POPULATION;p++){
			for(int t=0;t<FACILITY;t++){
		    facility[t] = t;
			}
			shuffle(facility);
			// for(int t=0;t<FACILITY;t++){
			// 	printf("%d ", facility[t]);
			// }
			for(int f=0;f<FACILITY;f++){
				data[i*POPULATION*FACILITY+p*FACILITY+f] = facility[f];
				printf("%d ", data[i*POPULATION*FACILITY+p*FACILITY+f]);
			}
			printf("\n");
			for(int b=0;b<FACILITY-1;b++){
				int j = rand() % 2;
		    bay[i*POPULATION*FACILITY+p*(FACILITY-1)+b] = j;
			}
		}
	}

  printf("data\n");
	for(int i=0;i<ISLAND;i++){
		for(int p=0;p<POPULATION;p++){
			for(int f=0;f<FACILITY;f++){
				printf("%d ", data[i*POPULATION*FACILITY+p*FACILITY+f]);
			}
			printf("\n");
		}
		printf("\n");
	}

  printf("bay\n");
	for(int i=0;i<ISLAND;i++){
		for(int p=0;p<POPULATION;p++){
			for(int f=0;f<FACILITY-1;f++){
				printf("%d ", bay[i*POPULATION*FACILITY+p*(FACILITY-1)+f]);
			}
			printf("\n");
		}
		printf("\n");
	}

  int *GA;
  short int *GB;
  hipMalloc((void**)&GA, ISLAND*POPULATION*FACILITY*sizeof(int));
	hipMemcpy(GA, data, ISLAND*POPULATION*FACILITY*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&GB, ISLAND*POPULATION*(FACILITY-1)*sizeof(short int));
	hipMemcpy(GB, bay, ISLAND*POPULATION*(FACILITY-1)*sizeof(short int), hipMemcpyHostToDevice);

  // read ther cost
	FILE *fPtr;

	fPtr=fopen("cost.txt","r");
	int cost[FACILITY*FACILITY] = {0};
	int temp[15*3]; // cost
	for(int i=0;i<15;i++){
		for(int a=0;a<3;a++){
			fscanf(fPtr , "%d " , &temp[i*3 + a]);
		}
	}
	fclose(fPtr);
	for(int i=0;i<15;i++){ // 2 dimention cost
		cost[ (temp[i*3]-1)*FACILITY + temp[i*3+1]-1] = temp[ i*3 + 2];
	}
  printf("cost: \n");
  for(int i=0;i<FACILITY;i++){ // 2 dimention cost
    for(int j=0;j<FACILITY;j++){
      printf("%d ", cost[i*FACILITY + j]);
    }
    printf("\n");
	}
  int *Gcost;
  hipMalloc((void**)&Gcost, FACILITY*FACILITY*sizeof(int));
  hipMemcpy(Gcost, cost, FACILITY*FACILITY*sizeof(int), hipMemcpyHostToDevice);


  float *Gposition;
  hipMalloc((void**)&Gposition, ISLAND*POPULATION*FACILITY*2*sizeof(float));

  // int *Gposition2;
  // cudaMalloc((void**)&Gposition2, ISLAND*POPULATION*FACILITY*2*sizeof(int));

  int g=ISLAND, b=POPULATION;
  // int m=g*b;
  calPosition<<<g, b>>>(GA, GB, Gposition);

  float position[ISLAND*POPULATION*FACILITY*2];
  // int position2[ISLAND*POPULATION*FACILITY*2];

  int data2[ISLAND*POPULATION*FACILITY];
  short int bay2[ISLAND*POPULATION*(FACILITY-1)]; //bay

  hipMemcpy(data2, GA, ISLAND*POPULATION*FACILITY*sizeof(int), hipMemcpyDeviceToHost);

  printf("data2\n");
	for(int i=0;i<ISLAND;i++){
		for(int p=0;p<POPULATION;p++){
			for(int f=0;f<FACILITY;f++){
				printf("%d ", data[i*POPULATION*FACILITY+p*FACILITY+f]);
			}
			printf("\n");
		}
		printf("\n");
	}
  hipMemcpy(bay2, GB, ISLAND*POPULATION*(FACILITY-1)*sizeof(short int), hipMemcpyDeviceToHost);
  printf("bay2\n");
	for(int i=0;i<ISLAND;i++){
		for(int p=0;p<POPULATION;p++){
			for(int f=0;f<FACILITY-1;f++){
				printf("%d ", bay2[i*POPULATION*FACILITY+p*(FACILITY-1)+f]);
			}
			printf("\n");
		}
		printf("\n");
	}

  hipMemcpy(position, Gposition, ISLAND*POPULATION*FACILITY*2*sizeof(float), hipMemcpyDeviceToHost);

  // print position
	for(int i=0;i<ISLAND;i++){
		printf("island%d \n", i);
		for(int p=0;p<POPULATION;p++){
			printf("po%d = \n",p);
			for(int f=0;f<FACILITY;f++){
				for(int k=0;k<2;k++){
					printf("%f ", position[i*POPULATION*FACILITY*2+p*FACILITY*2+f*2+k]);
				}
				printf("\n");
			}
		}
	}

  for(int i=0;i<ISLAND*POPULATION*FACILITY*2;i++){
    printf("%f ", position[i]);
  }
  printf("\n");

  float distance[ISLAND*POPULATION*FACILITY*FACILITY] = {0};

  float *Gdistance;
  hipMalloc((void**)&Gdistance, ISLAND*POPULATION*FACILITY*FACILITY*sizeof(float));


  calDistance<<<g, b>>>(GA, Gposition, Gdistance);

	hipMemcpy(distance, Gdistance, ISLAND*POPULATION*FACILITY*FACILITY*sizeof(float), hipMemcpyDeviceToHost);

  printf("\ncalculate distance end\n");

  // print distance
	for(int i=0;i<ISLAND;i++){
		for(int p=0;p<POPULATION;p++){
      printf("po%d: \n", p);
			for(int f=0;f<FACILITY;f++){
				for(int j=0;j<FACILITY;j++){
					printf("%f ", distance[ i*POPULATION*FACILITY*FACILITY + p*FACILITY*FACILITY + f*FACILITY + j ]);
				}
				printf("\n");
			}
		}
	}


  float totalCost[ISLAND*POPULATION*FACILITY*FACILITY] = {0.0};

  float *GtotalCost;
  hipMalloc((void**)&GtotalCost, ISLAND*POPULATION*FACILITY*FACILITY*sizeof(float));

  calTotalcost<<<g, b>>>(Gdistance, Gcost, GtotalCost);

  hipMemcpy(totalCost, GtotalCost, ISLAND*POPULATION*FACILITY*FACILITY*sizeof(float), hipMemcpyDeviceToHost);

  // print totalCost
	for(int i=0;i<ISLAND;i++){
		for(int p=0;p<POPULATION;p++){
      printf("po%d: \n", p);
			for(int f=0;f<FACILITY;f++){
				for(int j=0;j<FACILITY;j++){
					printf("%f ", totalCost[i*POPULATION*FACILITY*FACILITY + p*FACILITY*FACILITY + f*FACILITY + j]);
				}
				printf("\n");
			}
		}
	}



  hipFree(Gdistance);
  hipFree(Gposition);
  return 0;

}

void shuffle(int* facility) { // ���ñƧǦn��facility
    int i;
    for(i = 0; i < FACILITY; i++) {
        int j = rand() % FACILITY;
        int tmp = facility[i];
        facility[i] = facility[j];
        facility[j] = tmp;
    }
}
