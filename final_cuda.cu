#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>


#define ISLAND 10
#define POPULATION 50
#define FACILITY 20
#define GENERATION 70
#define CROSSOVER 0.6
#define MUTATION 0.03
#define MIGRATION 15
#define INDIVIDUAL 5

#define H 15 // BAY height
#define W 10 // BAY width

void shuffle(short* facility);

__global__ void calPosition(short *data, bool *bay, float *position){

  short b=blockIdx.x;       //區塊索引 == ISLAND
  short t=threadIdx.x;      //執行緒索引 == POPULATION
  short n=blockDim.x;       //區塊中包含的執行緒數目 == num of ISLAND
  short x=b*n+t;

  short posit = x * FACILITY;
  short bayposit = x * (FACILITY-1);
  // int posit=b*POPULATION*FACILITY+t*FACILITY;            //執行緒在陣列中對應的位置
  // int posofposit = b*POPULATION*FACILITY*2+t*FACILITY*2;

  for(int i=0;i<ISLAND*POPULATION*FACILITY*2;i++){
    position[i] = 0;
  }


			short len = 1;
			short next = 0;
			for(short f=0;f<FACILITY;f++){
				if(bay[bayposit+f] == 0){
					len = len + 1;
				}
				if(bay[bayposit+f] == 1 || f == FACILITY - 1 ){
					if(f == FACILITY - 1 && bay[bayposit+f] == 0){
						len = len - 1;
					}
					float x = W / 2.0 + next;

					for(short j=0;j<len;j++){

						position[posit*2+(f+j-len+1)*2] = x;

						float y = H / (len * 2.0) * ( (j * 2) + 1) ;

						position[posit*2+(f+j-len+1)*2+1] = y;
					}
					len = 1;

					next = next + W;
				}
			}
}

__global__ void calDistance(short *data, float *position, float *distance){

  short b=blockIdx.x;       //區塊索引 == ISLAND
  short t=threadIdx.x;      //執行緒索引 == POPULATION
  short n=blockDim.x;       //區塊中包含的執行緒數目 == num of ISLAND
  short x=b*n+t;

  short posit = x * FACILITY;

  // int posit=b*POPULATION*FACILITY+t*FACILITY;            //執行緒在陣列中對應的位置
  // int posofposit = b*POPULATION*FACILITY*2+t*FACILITY*2;

  for(int i=0;i<ISLAND*POPULATION*FACILITY*FACILITY;i++){
    distance[i] = 0;
  }


  for(short f=0;f<FACILITY;f++){
    // printf("\ndistance calculate facility%d\n", f);
    for(short j=f+1;j<FACILITY;j++){

      float x1 = position[ (posit + f)*2 ];
      float y1 = position[ (posit + f)*2 + 1];

      short x = data[ posit + f ];
      // printf("x = %d\n", x);
      float x2 = position[ (posit + j)*2 ];
      float y2 = position[ (posit + j)*2 + 1];
      short y = data[ posit + j ];
      // printf("y= %d\n", y);
      if(y2 > y1){
        distance[ (posit + x)*FACILITY + y] = sqrt( (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) ) ;
        distance[ (posit + y)*FACILITY + x] = sqrt( (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) ) ;
      }
      else{
        distance[ (posit + x)*FACILITY + y] = sqrt( (x2 - x1) * (x2 - x1) + (y1 - y2) * (y1 - y2) ) ;
        distance[ (posit + y)*FACILITY + x] = sqrt( (x2 - x1) * (x2 - x1) + (y1 - y2) * (y1 - y2) ) ;
      }
    }
  }

}

__global__ void calTotalcost(float *distance, int *cost, float *totalCost){

  short b=blockIdx.x;       //區塊索引 == ISLAND
  short t=threadIdx.x;      //執行緒索引 == POPULATION
  short n=blockDim.x;       //區塊中包含的執行緒數目 == num of ISLAND
  short x=b*n+t;

  short posit = x * FACILITY;

  // short posit=b*POPULATION*FACILITY+t*FACILITY;            //執行緒在陣列中對應的位置
  // int posofposit = b*POPULATION*FACILITY*2+t*FACILITY*2;

  // for(int i=0;i<ISLAND*POPULATION*FACILITY*FACILITY;i++){
  //   totalCost[i] = 0;
  // }


	for(short f=0;f<FACILITY;f++){
		for(short j=0;j<FACILITY;j++){
			totalCost[ (posit + f)*FACILITY + j] = cost[f*FACILITY + j] * distance[ (posit + f)*FACILITY + j];
		}
	}

}

__global__ void calOF(float *sumCost, float *minCost, float *totalCost){

  short b=blockIdx.x;       //區塊索引 == ISLAND
  short t=threadIdx.x;      //執行緒索引 == POPULATION
  short n=blockDim.x;       //區塊中包含的執行緒數目 == num of ISLAND
  short x=b*n+t;

  short posit = x * FACILITY;

  // calculate OF

	sumCost[x] = 0.0;
	minCost[x/POPULATION * 2] = 0.0;


			for(short f=0;f<FACILITY;f++){
				for(short j=0;j<FACILITY;j++){
					sumCost[x] += totalCost[ (posit + f)*FACILITY + j];
				}
			}
			if(x % POPULATION==0){
				minCost[(x/POPULATION)*2] = sumCost[x*FACILITY + 0];
				minCost[(x/POPULATION)*2 + 1] = 0;
			}else if(minCost[x/POPULATION] > sumCost[x]){
				minCost[(x/POPULATION)*2] = sumCost[x];
				minCost[(x/POPULATION)*2 + 1] = x % POPULATION;
			}


}


__global__ void calProbability(float *probability, float *totalPro, float *sumCost){

  short b=blockIdx.x;       //區塊索引 == ISLAND
  short t=threadIdx.x;      //執行緒索引 == POPULATION
  short n=blockDim.x;       //區塊中包含的執行緒數目 == num of ISLAND
  short x=b*n+t;


  probability[x] = (1.0 / sumCost[x]) / (totalPro[ x / POPULATION ]) ;

}


__global__ void crossOver(float *probability2, short *data, bool *bay, short *data2, bool *bay2, int *tem, int *tem2, int *Gyes, int *Gsss, int *Gcount, int *GGgetP, int *GGgetP2, float *test){

  short b=blockIdx.x;       //區塊索引 == ISLAND
  short t=threadIdx.x;      //執行緒索引 == POPULATION
  short n=blockDim.x;       //區塊中包含的執行緒數目 == num of ISLAND
  short x=b*n+t;

  short posit = x * 2 * FACILITY;
  short posit2 = (2*x+1) * FACILITY;
  short bayposit = x * 2 * (FACILITY-1);
  short bayposit2 = (2*x+1) * (FACILITY-1);

			float get = (tem[x] % 10000) * 0.0001;
      test[x] = 0.0;
      tem[x] = tem[x] % 10000;
			short getP = 0;
			float get2 = tem2[x] % 10000 * 0.0001;
      tem2[x] = tem2[x] % 10000;
			short getP2 = 0;
      GGgetP2[x] = -1;
			for(short p=0;p<POPULATION-1;p++){
				if(get >= probability2[ (x/POPULATION)*POPULATION + p ] && get < probability2[ (x/POPULATION)*POPULATION + p+1 ]){
					getP = p+1;
          GGgetP[x] = (x/POPULATION)*POPULATION + p;
					break;
				}
				else if(p==POPULATION-2){
					getP = p+1;
          GGgetP[x] = (x/POPULATION)*POPULATION + p;
					break;
				}
			}
      test[x] = probability2[ (x/POPULATION)*POPULATION + 1];

			for(short p=0;p<POPULATION-1;p++){
				if(get2 >= probability2[x/POPULATION*POPULATION + p] && get2 < probability2[x/POPULATION*POPULATION + p+1]){
					getP2 = p+1;
          GGgetP2[x] = (x/POPULATION)*POPULATION + p;
					break;
				}
				else if(p==POPULATION-2){
					getP2 = p+1;
          GGgetP2[x] = (x/POPULATION)*POPULATION + p;
					break;
				}
			}

			for(short f=0;f<FACILITY;f++){
				data2[ posit + f] = data[ x/POPULATION*POPULATION*FACILITY + getP*FACILITY + f];
				bay2[ (2 * x)*(FACILITY-1) + f] = bay[ x/POPULATION*POPULATION*(FACILITY-1) + getP*(FACILITY-1) + f];
			}




			for(short f=0;f<FACILITY;f++){
				data2[ posit2 + f ] = data[x/POPULATION*POPULATION*FACILITY + getP2*FACILITY + f];
				bay2[ (2 * x + 1)*(FACILITY-1) + f] = bay[x/POPULATION*POPULATION*(FACILITY-1) + getP2*(FACILITY-1) + f];
			}


      int tt = Gyes[x] % 10000;
			float yes = tt * 0.0001;
      Gyes[x] = tt;

			if(yes <= CROSSOVER){

				short sss = FACILITY - 1;
        int seq = Gsss[x] % sss;
        Gsss[x] = seq;


				int cross[4][2];

				cross[0][0] = data2[ posit + seq];
				cross[0][1] = data2[ posit2 + seq];
				cross[1][0] = data2[ posit + seq];
				cross[1][1] = data2[ posit2 + seq+1];
				cross[2][0] = data2[ posit + seq+1];
				cross[2][1] = data2[ posit2 + seq];
				cross[3][0] = data2[ posit+ seq+1];
				cross[3][1] = data2[ posit2 + seq+1];



				short temp = data2[ posit2 + seq];

				short temp2 = data2[posit2 + seq+1];

				data2[ posit2 + seq] = data2[ posit + seq];

				data2[ posit2 + seq+1] = data2[posit + seq+1];

				data2[posit + seq] = temp;
				data2[posit + seq+1] = temp2;



				short count = 0;
				for(short c=0;c<4;c++){
					if(cross[c][0] == cross[c][1]){
						count++;
					}
				}
        Gcount[x] = count;

				switch (count) {
					case 0:
						for(short c=0;c<FACILITY;c++){
							if(c != seq){
								if(data2[posit + c] == cross[0][1]){
									data2[ posit + c] = cross[0][0];
								}
								if(data2[posit + c] == cross[3][1]){
									data2[ posit + c] = cross[3][0];
								}
							}
							else{
								c++;
							}
						}

						for(short c=0;c<FACILITY;c++){
							if(c != seq){
								if(data2[posit2 + c] == cross[0][0]){
									data2[ posit2 + c] = cross[0][1];
								}
								if(data2[posit2 + c] == cross[3][0]){
									data2[ posit2 + c] = cross[3][1];
								}
							}
							else{
								c++;
							}
						}
						break;
					case 1:
						temp = 99;
						for(short c=0;c<4;c++){
							if(cross[c][0] == cross[c][1]){
								temp = cross[c][0];
							}
						}

						for(short c=0;c<4;c++){
							if(cross[c][0] != temp && cross[c][1] != temp){
								for(short f=0;f<FACILITY;f++){
									if(f != seq){
										if(data2[posit + f] == cross[c][1]){
											data2[ posit + f] = cross[c][0];
										}
									}
									else{
										f++;
									}
								}
							}
						}

						for(short c=0;c<4;c++){
							if(cross[c][0] != temp && cross[c][1] != temp){
								for(short f=0;f<FACILITY;f++){
									if(f != seq){
										if(data2[posit2 + f] == cross[c][0]){
											data2[ posit2 + f] = cross[c][1];
										}
									}
									else{
										f++;
									}
								}
							}
						}
						break;
					case 2:
						break;
				}




				temp = bay2[bayposit2 + seq];
				temp2 = bay2[bayposit2 + seq+1];
				bay2[bayposit2 + seq]   = bay2[bayposit + seq];
				bay2[bayposit2 + seq+1] = bay2[bayposit + seq+1];
				bay2[bayposit + seq]   = bay2[bayposit2 + seq];
				bay2[bayposit + seq+1] = bay2[bayposit2 + seq+1];
			}else {

			}



}

__global__ void mutation(short *data2, int *mutaYes, int *mutaTem, int *mutaTem2){
  int b=blockIdx.x;       //區塊索引 == ISLAND
  int t=threadIdx.x;      //執行緒索引 == POPULATION
  int n=blockDim.x;       //區塊中包含的執行緒數目 == num of ISLAND
  int x=b*n+t;

  int posit = x * FACILITY;

  float yes = (mutaYes[x] % 10000) * 0.0001;
	// fprintf(FIN, "取得%f \n", yes);
	if(yes < MUTATION){
		// fprintf(FIN, "第%d突變\n", p);
		short get = mutaTem[x] % FACILITY;
		short get2 = mutaTem2[x] % FACILITY;
		short temp = data2[posit + get];
		data2[posit + get] = data2[posit + get2];
		data2[posit + get2] = temp;
	}else {
	}
}

__global__ void mutationBay(bool *bay2, int *mutaBayYes, int *mutaBayTem){
  short b=blockIdx.x;       //區塊索引 == ISLAND
  short t=threadIdx.x;      //執行緒索引 == POPULATION
  short n=blockDim.x;       //區塊中包含的執行緒數目 == num of ISLAND
  short x=b*n+t;

  short posit = x * (FACILITY - 1);

	float yes = mutaBayYes[x] % 10000 * 0.0001 ;
	if(yes < MUTATION){
		short get = mutaBayTem[x] % (FACILITY - 1);
		if(bay2[posit + get] == 0){
			bay2[posit + get] = 1;
		}else {
			bay2[posit + get] = 0;
		}
	}
}

int main(){

  double START,END;
  START = clock();
  srand(time(NULL));

  short data[ISLAND][POPULATION][FACILITY];
  bool bay[ISLAND][POPULATION][FACILITY-1]; //bay

  short facility[FACILITY];

  for(short i=0;i<ISLAND;i++){ // shuffle the sorted facility
		// printf("new island%d\n", i);
		for(short p=0;p<POPULATION;p++){
			for(short t=0;t<FACILITY;t++){
		    facility[t] = t;
			}
			shuffle(facility);
			// for(int t=0;t<FACILITY;t++){
			// 	printf("%d ", facility[t]);
			// }
			for(short f=0;f<FACILITY;f++){
				data[i][p][f] = facility[f];
				// printf("%d ", data[i][p][f]);
			}
			// printf("\n");
			for(short b=0;b<FACILITY-1;b++){
				bool j = rand() % 2;
		    bay[i][p][b] = j;
			}
		}
	}

  // printf("data\n");
	// for(int i=0;i<ISLAND;i++){
	// 	for(int p=0;p<POPULATION;p++){
	// 		for(int f=0;f<FACILITY;f++){
	// 			printf("%d ", data[i][p][f]);
	// 		}
	// 		printf("\n");
	// 	}
	// 	printf("\n");
	// }

  // printf("bay\n");
	// for(int i=0;i<ISLAND;i++){
	// 	for(int p=0;p<POPULATION;p++){
	// 		for(int f=0;f<FACILITY-1;f++){
	// 			printf("%d ", bay[i][p][f]);
	// 		}
	// 		printf("\n");
	// 	}
	// 	printf("\n");
	// }

  // int *GA;
  // short int *GB;
  // hipMalloc((void**)&GA, ISLAND*POPULATION*FACILITY*sizeof(int));
	// hipMemcpy(GA, data, ISLAND*POPULATION*FACILITY*sizeof(int), hipMemcpyHostToDevice);
  //
	// hipMalloc((void**)&GB, ISLAND*POPULATION*(FACILITY-1)*sizeof(short int));
	// hipMemcpy(GB, bay, ISLAND*POPULATION*(FACILITY-1)*sizeof(short int), hipMemcpyHostToDevice);

  // read ther cost
	FILE *fPtr;

  short ttt = FACILITY * (FACILITY-1) ;

	fPtr=fopen("cost.txt","r");
	short cost[FACILITY][FACILITY] = {0};
	short temp[ttt][3]; // cost
	for(short i=0;i<ttt;i++){
		fscanf(fPtr , "%d %d %d" , &temp[i][0], &temp[i][1], &temp[i][2]);
	}
	fclose(fPtr);
	for(short i=0;i<ttt;i++){ // 2 dimention cost
		cost[ temp[i][0]-1 ][ temp[i][1]-1] = temp[i][2];
	}
  // printf("cost: \n");
  // for(int i=0;i<FACILITY;i++){ // 2 dimention cost
  //   for(int j=0;j<FACILITY;j++){
  //     printf("%d ", cost[i][j]);
  //   }
  //   printf("\n");
	// }
  short *Gcost;
  hipMalloc((void**)&Gcost, FACILITY*FACILITY*sizeof(short));
  hipMemcpy(Gcost, cost, FACILITY*FACILITY*sizeof(short), hipMemcpyHostToDevice);


  for(short gggggg=0;gggggg<GENERATION;gggggg++){ // generation

  // printf("\n*****%d的generation*****\n", gggggg);
  short *GA;
  bool *GB;
  hipMalloc((void**)&GA, ISLAND*POPULATION*FACILITY*sizeof(short));
	hipMemcpy(GA, data, ISLAND*POPULATION*FACILITY*sizeof(short), hipMemcpyHostToDevice);

	hipMalloc((void**)&GB, ISLAND*POPULATION*(FACILITY-1)*sizeof(bool));
	hipMemcpy(GB, bay, ISLAND*POPULATION*(FACILITY-1)*sizeof(bool), hipMemcpyHostToDevice);


  float *Gposition;
  hipMalloc((void**)&Gposition, ISLAND*POPULATION*FACILITY*2*sizeof(float));

  // int *Gposition2;
  // hipMalloc((void**)&Gposition2, ISLAND*POPULATION*FACILITY*2*sizeof(int));

  short g=ISLAND, b=POPULATION;
  // int m=g*b;
  calPosition<<<g, b>>>(GA, GB, Gposition);

  float position[ISLAND*POPULATION*FACILITY*2];
  // int position2[ISLAND*POPULATION*FACILITY*2];

  // int data2[ISLAND*POPULATION*FACILITY];
  // short int bay2[ISLAND*POPULATION*(FACILITY-1)]; //bay
  //
  // hipMemcpy(data2, GA, ISLAND*POPULATION*FACILITY*sizeof(int), hipMemcpyDeviceToHost);
  //
  // printf("data2\n");
	// for(int i=0;i<ISLAND;i++){
	// 	for(int p=0;p<POPULATION;p++){
	// 		for(int f=0;f<FACILITY;f++){
	// 			printf("%d ", data[i*POPULATION*FACILITY+p*FACILITY+f]);
	// 		}
	// 		printf("\n");
	// 	}
	// 	printf("\n");
	// }
  // hipMemcpy(bay2, GB, ISLAND*POPULATION*(FACILITY-1)*sizeof(short int), hipMemcpyDeviceToHost);
  // printf("bay2\n");
	// for(int i=0;i<ISLAND;i++){
	// 	for(int p=0;p<POPULATION;p++){
	// 		for(int f=0;f<FACILITY-1;f++){
	// 			printf("%d ", bay2[i*POPULATION*FACILITY+p*(FACILITY-1)+f]);
	// 		}
	// 		printf("\n");
	// 	}
	// 	printf("\n");
	// }

  hipMemcpy(position, Gposition, ISLAND*POPULATION*FACILITY*2*sizeof(float), hipMemcpyDeviceToHost);

  // print position
	// for(int i=0;i<ISLAND;i++){
	// 	printf("island%d \n", i);
	// 	for(int p=0;p<POPULATION;p++){
	// 		printf("po%d = \n",p);
	// 		for(int f=0;f<FACILITY;f++){
	// 			for(int k=0;k<2;k++){
	// 				printf("%f ", position[i*POPULATION*FACILITY*2+p*FACILITY*2+f*2+k]);
	// 			}
	// 			printf("\n");
	// 		}
	// 	}
	// }

  // for(int i=0;i<ISLAND*POPULATION*FACILITY*2;i++){
  //   printf("%f ", position[i]);
  // }
  // printf("\n");

  float distance[ISLAND*POPULATION*FACILITY*FACILITY] = {0};

  float *Gdistance;
  hipMalloc((void**)&Gdistance, ISLAND*POPULATION*FACILITY*FACILITY*sizeof(float));


  calDistance<<<g, b>>>(GA, Gposition, Gdistance);

	hipMemcpy(distance, Gdistance, ISLAND*POPULATION*FACILITY*FACILITY*sizeof(float), hipMemcpyDeviceToHost);

  // printf("\ncalculate distance end\n");

  // print distance
	// for(int i=0;i<ISLAND;i++){
	// 	for(int p=0;p<POPULATION;p++){
  //     printf("po%d: \n", p);
	// 		for(int f=0;f<FACILITY;f++){
	// 			for(int j=0;j<FACILITY;j++){
	// 				printf("%f ", distance[ i*POPULATION*FACILITY*FACILITY + p*FACILITY*FACILITY + f*FACILITY + j ]);
	// 			}
	// 			printf("\n");
	// 		}
	// 	}
	// }


  float totalCost[ISLAND][POPULATION][FACILITY][FACILITY] = {0.0};

  float *GtotalCost;
  hipMalloc((void**)&GtotalCost, ISLAND*POPULATION*FACILITY*FACILITY*sizeof(float));

  calTotalcost<<<g, b>>>(Gdistance, Gcost, GtotalCost);

  hipMemcpy(totalCost, GtotalCost, ISLAND*POPULATION*FACILITY*FACILITY*sizeof(float), hipMemcpyDeviceToHost);

  // print totalCost
	// for(int i=0;i<ISLAND;i++){
	// 	for(int p=0;p<POPULATION;p++){
  //     printf("po%d: \n", p);
	// 		for(int f=0;f<FACILITY;f++){
	// 			for(int j=0;j<FACILITY;j++){
	// 				printf("%f ", totalCost[i][p][f][j]);
	// 			}
	// 			printf("\n");
	// 		}
	// 	}
	// }

  float *GsumCost;
  float sumCost[ISLAND][POPULATION]={0.0};

  hipMalloc((void**)&GsumCost, ISLAND*POPULATION*sizeof(float));

  float *GminCost;
  float minCost[ISLAND][2];
  hipMalloc((void**)&GminCost, ISLAND*2*sizeof(float));

  calOF<<<g, b>>>(GsumCost, GminCost, GtotalCost);

  hipMemcpy(sumCost, GsumCost, ISLAND*POPULATION*sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(minCost, GminCost, ISLAND*2*sizeof(float), hipMemcpyDeviceToHost);

  // printf("\n");
	// for(int i=0;i<ISLAND;i++){
	// 	printf("第%d島嶼: \n", i);
	// 	for(int p=0;p<POPULATION;p++){
	// 		printf("%d: ", p);
	// 	  printf("sum = %f", sumCost[i][p]);
	// 		printf("\n");
	// 	}
	// }


  short data2[ISLAND][POPULATION][FACILITY]; // facility
  short *Gdata2;
  hipMalloc((void**)&Gdata2, ISLAND*POPULATION*FACILITY*sizeof(short));
	bool bay2[ISLAND][POPULATION][FACILITY-1]; //bay
  bool *Gbay2;
  hipMalloc((void**)&Gbay2, ISLAND*POPULATION*(FACILITY-1)*sizeof(bool));

	float probability[ISLAND][POPULATION] = {0.0}; // �U�Ӿ��v

  // for(int i=0;i<ISLAND;i++){
	// 	for(int p=0;p<POPULATION;p++){
	// 		printf("pro%f \n", probability[i][p]);
	// 	}
	// }
  float *Gprobability;
  hipMalloc((void**)&Gprobability, ISLAND*POPULATION*sizeof(float));

	float totalPro[ISLAND] = {0.0};                // �`(�����˼�)
  float *GtotalPro;
  hipMalloc((void**)&GtotalPro, ISLAND*sizeof(float));

  for(short i=0;i<ISLAND;i++){
		for(short p=0;p<POPULATION;p++){
			totalPro[i] = totalPro[i] + (1.0 / sumCost[i][p]);
			// printf("%f %f\n", totalPro[i], (1.0 / sumCost[i][p]));
		}
	}

  hipMemcpy(GtotalPro, totalPro, ISLAND*sizeof(float), hipMemcpyHostToDevice);


	// for(int i=0;i<ISLAND;i++){
	// 	for(int p=0;p<POPULATION;p++){
	// 		printf("%f %f\n", totalPro[i], (1.0 / sumCost[i][p]));
	// 	}
	// }

  calProbability<<<ISLAND, POPULATION>>>(Gprobability, GtotalPro, GsumCost);

  hipMemcpy(probability, Gprobability, ISLAND*POPULATION*sizeof(float), hipMemcpyDeviceToHost);

	// for(int i=0;i<ISLAND;i++){
  //   printf("\n");
	// 	for(int p=0;p<POPULATION;p++){
	// 		printf("%f %f %f \n", probability[i][p], (1.0 / sumCost[i][p]), totalPro[i]);
	// 	}
	// }


  float probability2[ISLAND][POPULATION] = {0.0};
	for(short i=0;i<ISLAND;i++){
		for(short p=0;p<POPULATION;p++){
			for(short j=0;j<=p;j++){
				probability2[i][p] += probability[i][j];
			}
		}
	}

  float *Gprobability2;
  hipMalloc((void**)&Gprobability2, ISLAND*POPULATION*sizeof(float));
  hipMemcpy(Gprobability2, probability2, ISLAND*POPULATION*sizeof(float), hipMemcpyHostToDevice);

	// print probability2 (Roulette)
	// printf("probability2\n");
	// for(int i=0;i<ISLAND;i++){
	// 	for(int p=0;p<POPULATION;p++){
	// 		printf("%f ", probability2[i][p]);
	// 	}
	// }



  int *Gtem, *Gtem2, *Gyes, *Gsss;// choose two to crossover and if yes or not and choose area
  int tem[ISLAND*POPULATION], tem2[ISLAND*POPULATION], yes[ISLAND*POPULATION], sss[ISLAND*POPULATION];
  hipMalloc((void**)&Gtem, ISLAND*POPULATION*sizeof(int));
  hipMalloc((void**)&Gtem2, ISLAND*POPULATION*sizeof(int));
  hipMalloc((void**)&Gyes, ISLAND*POPULATION*sizeof(int));
  hipMalloc((void**)&Gsss, ISLAND*POPULATION*sizeof(int));

  int *GmutaYes, *GmutaTem, *GmutaTem2;
  int mutaYes[ISLAND*POPULATION], mutaTem[ISLAND*POPULATION], mutaTem2[ISLAND*POPULATION];
  hipMalloc((void**)&GmutaYes, ISLAND*POPULATION*sizeof(int));
  hipMalloc((void**)&GmutaTem, ISLAND*POPULATION*sizeof(int));
  hipMalloc((void**)&GmutaTem2, ISLAND*POPULATION*sizeof(int));
  for(int i=0;i<ISLAND*POPULATION;i++){
    tem[i] = rand(); // first change
    tem2[i] = rand(); // second change
    yes[i] = rand(); // crossover or not
    sss[i] = rand(); // bay to crossover
    mutaYes[i] = rand(); // mutation or not
    mutaTem[i] = rand(); // first to change
    mutaTem2[i] = rand(); // second to change
    // printf("%d %d %d %d %d %d %d\n", tem[i], tem2[i], yes[i], sss[i], mutaYes[i], mutaTem[i], mutaTem2[i]);
  }

  hipMemcpy(Gtem, tem, ISLAND*POPULATION*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Gtem2, tem2, ISLAND*POPULATION*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Gyes, yes, ISLAND*POPULATION*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(Gsss, sss, ISLAND*POPULATION*sizeof(int), hipMemcpyHostToDevice);


  hipMemcpy(GmutaYes, mutaYes, ISLAND*POPULATION*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(GmutaTem, mutaTem, ISLAND*POPULATION*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(GmutaTem2, mutaTem2, ISLAND*POPULATION*sizeof(int), hipMemcpyHostToDevice);

  int *Gcount;
  hipMalloc((void**)&Gcount, ISLAND*POPULATION*sizeof(int));
  int *GetP, *GetP2;
  hipMalloc((void**)&GetP, ISLAND*POPULATION*sizeof(int));
  hipMalloc((void**)&GetP2, ISLAND*POPULATION*sizeof(int));
  int getP[ISLAND*POPULATION], getP2[ISLAND*POPULATION];
  float *Gtest;
  hipMalloc((void**)&Gtest, ISLAND*POPULATION*sizeof(float));
  float test[ISLAND*POPULATION] = {0.0};
  crossOver<<<ISLAND, POPULATION / 2>>>(Gprobability2, GA, GB, Gdata2, Gbay2, Gtem, Gtem2, Gyes, Gsss, Gcount, GetP, GetP2, Gtest);
  hipMemcpy(data2, Gdata2, ISLAND*POPULATION*FACILITY*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(bay2, Gbay2, ISLAND*POPULATION*(FACILITY-1)*sizeof(short int), hipMemcpyDeviceToHost);

  int count[ISLAND*POPULATION] = {0};
  hipMemcpy(tem, Gtem, ISLAND*POPULATION*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(tem2, Gtem2, ISLAND*POPULATION*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(yes, Gyes, ISLAND*POPULATION*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(sss, Gsss, ISLAND*POPULATION*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(count, Gcount, ISLAND*POPULATION*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(getP, GetP, ISLAND*POPULATION / 2*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(getP2, GetP2, ISLAND*POPULATION / 2*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(test, Gtest, ISLAND*POPULATION*sizeof(int), hipMemcpyDeviceToHost);

  // mutation facility
	// printf("\nready to mutation\n");

  mutation<<<ISLAND, POPULATION>>>(Gdata2, GmutaYes, GmutaTem, GmutaTem2);

  hipMemcpy(mutaYes, GmutaYes, ISLAND*POPULATION*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(mutaTem, GmutaTem, ISLAND*POPULATION*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(mutaTem2, GmutaTem2, ISLAND*POPULATION*sizeof(int), hipMemcpyDeviceToHost);

  int *GmutaBayYes, *GmutaBayTem;
  int mutaBayYes[ISLAND*POPULATION], mutaBayTem[ISLAND*POPULATION];
  hipMalloc((void**)&GmutaBayYes, ISLAND*POPULATION*sizeof(int));
  hipMalloc((void**)&GmutaBayTem, ISLAND*POPULATION*sizeof(int));

  hipMemcpy(GmutaBayYes, mutaBayYes, ISLAND*POPULATION*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(GmutaBayTem, mutaBayTem, ISLAND*POPULATION*sizeof(int), hipMemcpyHostToDevice);

  mutationBay<<<ISLAND, POPULATION>>>(Gbay2, GmutaBayYes, GmutaBayTem);

  hipMemcpy(mutaBayYes, GmutaBayYes, ISLAND*POPULATION*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(mutaBayTem, GmutaBayTem, ISLAND*POPULATION*sizeof(int), hipMemcpyDeviceToHost);

  // migration
	if( (gggggg+1) % MIGRATION == 0 && (gggggg+1) != 0 && ISLAND > 1){
		// printf("***migration***\n");

		int temp3[ISLAND][POPULATION/2][FACILITY];
		short temp4[ISLAND][POPULATION/2][FACILITY-1];
		int indexCost[ISLAND][POPULATION];

		for(int i=0;i<ISLAND;i++){
			for(int p=0;p<POPULATION;p++){
				indexCost[i][p] = p;
			}
		}

		// bubble sort
		// float temp;
		for(int k=0;k<ISLAND;k++){
			for(int i=POPULATION-1; i>=1; i--){
	      for(int j=0; j<=i-1; j++){
	        if(sumCost[k][j] > sumCost[k][j+1]){
							int temp2 = indexCost[k][j];
	            indexCost[k][j] = indexCost[k][j+1];
	            indexCost[k][j+1] = temp2;
	        }
	      }
	    }
		}

		// print sorted index
		// for(int i=0;i<ISLAND;i++){
		// 	for(int p=0;p<POPULATION;p++){
		// 		printf("%d ", indexCost[i][p]);
		// 	}
		// 	printf("\n");
		// }

		int countP = 0;
		for(int i=0;i<ISLAND;i++){
			while(countP < INDIVIDUAL){
				for(int p=0;p<POPULATION;p++){
					if(p == indexCost[i][countP]){
						for(int f=0;f<FACILITY;f++){
							temp3[i][countP][f] = data2[i][p][f];
						}
						for(int f=0;f<FACILITY-1;f++){
							temp4[i][countP][f] = bay2[i][p][f];
						}
						countP++;
						break;
					}
				} // population end
			}
			countP = 0;
		} // island end

		for(int i=0;i<ISLAND;i++){
			if(i==0){
				for(int k=0;k<POPULATION/2;k++){
					int backP = indexCost[ISLAND-1][k];
					int frontP = indexCost[i][k];
					for(int f=0;f<FACILITY;f++){
						data2[i][frontP][f] = temp3[ISLAND-1][backP][f];
					}
					for(int f=0;f<FACILITY-1;f++){
						bay2[i][frontP][f] = temp4[ISLAND-1][backP][f];
					}
				}
			}else {
				for(int k=0;k<POPULATION/2;k++){
					int backP = indexCost[i-1][k];
					int frontP = indexCost[i][k];
					// int p = indexCost[i][k];
					for(int f=0;f<FACILITY;f++){
						data2[i][frontP][f] = temp3[ISLAND-1][backP][f];
					}
					for(int f=0;f<FACILITY-1;f++){
						bay2[i][frontP][f] = temp4[ISLAND-1][backP][f];
					}
				}
			} // else end

		} // for end




		} // if migration end


  // printf("count: \n");
  // for(int i=0;i<10;i++){
  //   printf("%d ", count[i]);
  // }

  // printf("\nget: \n");
  // for(int i=0;i<ISLAND*POPULATION / 2;i++){
  //   printf("%d %d\n", getP[i], getP2[i]);
  // }

  // printf("\ntest: \n");
  // for(int i=0;i<10;i++){
  //   printf("%f\n", test[i]);
  // }

  // printf("\nTEM: \n");
  // for(int i=0;i<20;i++){
  //   printf("%d %d %d %d\n", tem[i], tem2[i], yes[i], sss[i]);
  // }
  //
  // printf("\nmutation: \n");
  // for(int i=0;i<20;i++){
  //   printf("%d %d %d\n", mutaYes[i], mutaTem[i], mutaTem2[i]);
  // }

  if(gggggg==69){
    int answerPos[2];
    float answer;
    answerPos[0] = 0;
    answerPos[1] = 0;
    answer = sumCost[0][0];
    for(int i=0;i<ISLAND;i++){
  		// printf("第%d島嶼(OF): \n", i);
  		for(int p=0;p<POPULATION;p++){
  			// printf("%f ", sumCost[i][p]);
        if(sumCost[i][p] < answer){
          answerPos[0] = i;
          answerPos[1] = p;
          answer = sumCost[i][p];
        }
  			// printf("\n");
  		}
  	}


    for(int i=0;i<FACILITY;i++){
      printf("%d ", data2[ answerPos[0] ][ answerPos[1] ][i]);
    }
    printf("\n");
    for(int i=0;i<FACILITY-1;i++){
      printf("%d ", bay2[ answerPos[0] ][ answerPos[1] ][i]);
    }
    printf("最小: %d %d = %f\n", answerPos[0], answerPos[1], answer);
  }



  // for(int i=0;i<ISLAND;i++){
  //   for(int p=0;p<POPULATION;p++){
  //     printf("\n交配結果(data2)%d\n", p);
  //     for(int f=0;f<FACILITY;f++){
  //       printf("%d ", data2[i][p][f]);
  //     }
  //     printf("\n");
  //   }
  // }

  // parent to child
	// printf("***chile to parent!!!***\n");
	for(int i=0;i<ISLAND;i++){
		// printf("island%d\n", i);
		for(int p=0;p<POPULATION;p++){
			for(int f=0;f<FACILITY;f++){
				data[i][p][f] = data2[i][p][f];
				// printf("%d ", data[i][p][f]);
			}
			// printf("\n");
		}
	}

  // 子代BAY
	for(int i=0;i<ISLAND;i++){
		for(int p=0;p<POPULATION;p++){
			for(int f=0;f<FACILITY-1;f++){
				bay[i][p][f] = bay2[i][p][f];
			}
		}
	}

  hipFree(GA);
  hipFree(GB);
  hipFree(Gdata2);
  hipFree(Gbay2);
  hipFree(GsumCost);
  hipFree(GminCost);
  hipFree(GtotalCost);
  hipFree(Gtem);
  hipFree(Gtem2);
  hipFree(GetP);
  hipFree(GetP2);
  hipFree(Gtest);
  hipFree(Gprobability);
  hipFree(Gprobability2);
  hipFree(Gyes);
  hipFree(Gsss);
  hipFree(GmutaYes);
  hipFree(GmutaTem);
  hipFree(GmutaTem2);
  hipFree(Gdistance);
  hipFree(Gposition);
  } // GENERATION 結束

  END = clock();
  printf("程式執行所花費： %lf S\n", (double)clock()/CLOCKS_PER_SEC);
  printf("進行運算所花費的時間： %lf S\n", (END - START) / CLOCKS_PER_SEC);
  // cout << endl << "程式執行所花費：" << (double)clock()/CLOCKS_PER_SEC << " S" ;
  // cout << endl << "進行運算所花費的時間：" << (END - START) / CLOCKS_PER_SEC << " S" << endl;
  return 0;

}

void shuffle(int* facility) { // ���ñƧǦn��facility
    int i;
    for(i = 0; i < FACILITY; i++) {
        int j = rand() % FACILITY;
        int tmp = facility[i];
        facility[i] = facility[j];
        facility[j] = tmp;
    }
}
