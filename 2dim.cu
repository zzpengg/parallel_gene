/*--------------------------------------------
* Date：2015-3-18
* Author：李根
* FileName：.cpp
* Description：CUDA二维数组加法
------------------------------------------------*/
#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>

static const int M = 4;
static const int N = 3;

//矩阵加法的kernel
__global__ void addMat(int **A,int **B,int **C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i < M && j < N)
        C[i][j] = A[i][j] + B[i][j];
}

int main()
{int **A = (int **)malloc(M*sizeof(int *));  //host memory
    int **B = (int **)malloc(M*sizeof(int *));  //host memory
    int **C = (int **)malloc(M*sizeof(int *));  //host memory
    int *dataA =(int *)malloc(M*N*sizeof(int )); //host memory data
    int *dataB = (int *)malloc(M*N*sizeof(int )); //host memory data
    int *dataC =(int *)malloc(M*N*sizeof(int )); //host memory data

    int **dev_A ;  //device memory
    int **dev_B ;  //device memory
    int **dev_C ;  //device memory
    int *dev_dataA ;  //device memory  data
    int *dev_dataB ;  //device memory  data
    int *dev_dataC ;  //device memory  data

    hipMalloc((void**)(&dev_A), M*sizeof(int*));
    hipMalloc((void**)(&dev_dataA), M*N*sizeof(int));
    hipMalloc((void**)(&dev_B), M*sizeof(int*));
    hipMalloc((void**)(&dev_dataB), M*N*sizeof(int));
    hipMalloc((void**)(&dev_C), M*sizeof(int*));
    hipMalloc((void**)(&dev_dataC), M*N*sizeof(int));

    for(int i=0;i<M*N;i++)
    {
        dataA[i] = i;
        dataB[i] = i+1;
        dataC[i] =0;
    }

    hipMemcpy((void*)(dev_dataA), (void*)(dataA), M*N*sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy((void*)(dev_dataB), (void*)(dataB), M*N*sizeof(int*), hipMemcpyHostToDevice);


    for(int i=0;i<M;i++)
    {
        A[i] = dev_dataA + N*i;
        B[i] = dev_dataB + N*i;
        C[i] = dev_dataC + N*i;
    }


    hipMemcpy((void*)(dev_A), (void*)(A), M*sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy((void*)(dev_B), (void*)(B), M*sizeof(int*), hipMemcpyHostToDevice);
    hipMemcpy((void*)(dev_C), (void*)(C), M*sizeof(int*), hipMemcpyHostToDevice);

    dim3 threadPerBlock(16,16);
    dim3 numBlocks((N+threadPerBlock.x-1)/(threadPerBlock.x), (M+threadPerBlock.y-1)/(threadPerBlock.y));
    addMat<<<numBlocks,threadPerBlock>>>(dev_A,dev_B,dev_C);
    hipMemcpy((void*)(dataC), (void*)(dev_dataC), M*N*sizeof(int), hipMemcpyDeviceToHost);
    for(int i=0;i<M*N;i++)
        std::cout<<dataC[i]<<" ";
    hipFree((void*)dev_dataC);
    hipFree((void*)dev_C);
    free(C);
    free(dataC);
    hipFree((void*)dev_dataB);
    hipFree((void*)dev_B);
    free(B);
    free(dataB);
    hipFree((void*)dev_dataA);
    hipFree((void*)dev_A);
    free(A);
    free(dataA);
    getchar();
}
